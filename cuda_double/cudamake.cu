#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include<hipblas.h>
#include<sys/time.h>
#include"cudamake.h"

int main(int argc, char *argv[])
{
        double *A, *B, *C_cuda, *C_csr, *C_cublas, *C_spmv, *val;
        int *row, *col;
       	double *A_d, *B_d, *C_cuda_d, *C_csr_d, *C_cublas_d, *C_spmv_d;
	int nnz;
        int N = atoi(argv[1]);
	double exe_time;
	struct timeval stop_time, start_time;
	
	mat_vec_allocation(N, &A, &B, &C_cuda, &C_csr, &C_cublas, &C_spmv, &val, &row, &col);
        
        sparse_mat_vec_initializer(N, A, B);
                
//        print_mat(N, A);
//        print_vec(N, B);

        double *val_d; 
        int *col_d, *row_d;
        
        hipMalloc(&A_d,N*N*sizeof(double));
        hipMalloc(&B_d,N*sizeof(double));
        hipMalloc(&C_cuda_d,N*sizeof(double));
        hipMalloc(&C_csr_d,N*sizeof(double));
        hipMalloc(&C_cublas_d,N*sizeof(double));
        hipMalloc(&C_spmv_d,N*sizeof(double));

        
        hipMemcpy(A_d,A,N*N*sizeof(double),hipMemcpyHostToDevice);
        hipMemcpy(B_d,B,N*sizeof(double),hipMemcpyHostToDevice);

	int Total_num_Threads = N;
        int num_threads_per_block = 256;
        int numblocks = Total_num_Threads/num_threads_per_block + 1;
        int  loop;
	//////////////////////////////////////////////////////////////////////
	exe_time = 0.0;
	for(loop = 0; loop < 5; loop++)
	{
	gettimeofday(&start_time, NULL);
	cuda_multiplication<<<numblocks,num_threads_per_block>>>(N, A_d,B_d,C_cuda_d);
	gettimeofday(&stop_time, NULL);
	exe_time += (stop_time.tv_sec+(stop_time.tv_usec/1000000.0)) - (start_time.tv_sec+(start_time.tv_usec/1000000.0));
	}
	printf("\nMatrix Size : %d X %d : Executed 5 times : Average execution time for naive mult kernel is = %lf seconds.\n", N,N, exe_time/5);
	hipMemcpy(C_cuda,C_cuda_d,N*sizeof(double),hipMemcpyDeviceToHost);
//	print_vec(C_cuda);
	///////////////////////////////////////////////////////////////////////
	///////////////////////////////////////////////////////////////////////
	
	nnz = mat_to_csr(N, A, val, row, col);
//	print_csr(N, val, row, col, nnz);
	
	hipMalloc(&val_d,nnz*sizeof(double));
        hipMalloc(&col_d,nnz*sizeof(int));
        hipMalloc(&row_d,(N+1)*sizeof(int));

        hipMemcpy(val_d,val,nnz*sizeof(double),hipMemcpyHostToDevice);
        hipMemcpy(col_d,col,nnz*sizeof(int),hipMemcpyHostToDevice);
        hipMemcpy(row_d,row,(N+1)*sizeof(int),hipMemcpyHostToDevice);
        
        exe_time = 0.0;
	for(loop = 0; loop < 5; loop++)
	{
        gettimeofday(&start_time, NULL);
	csr_multiplication<<<numblocks,num_threads_per_block>>>(N, B_d, C_csr_d, row_d, col_d, val_d);
	gettimeofday(&stop_time, NULL);
	exe_time += (stop_time.tv_sec+(stop_time.tv_usec/1000000.0)) - (start_time.tv_sec+(start_time.tv_usec/1000000.0));
	}
	printf("\nMatrix Size : %d X %d : Executed 5 times : Average execution time for csr mult kernel is = %lf seconds.\n", N,N, exe_time/5);
	hipMemcpy(C_csr, C_csr_d, N*sizeof(double), hipMemcpyDeviceToHost);
//	print_vec(C_csr);
	//////////////////////////////////////////////////////////////////////////
	/////////////////////////////////////////////////////////////////////////
	const double alpha = 1.0;
        const double beta = 0.0;

        hipblasHandle_t handle;
        hipblasCreate(&handle);
	
	exe_time = 0.0;
	for(loop = 0; loop < 5; loop++)
	{
	gettimeofday(&start_time, NULL);
        hipblasDgemv(handle, HIPBLAS_OP_T, N, N, &alpha, A_d, N, B_d, 1, &beta, C_cublas_d, 1);\
        gettimeofday(&stop_time, NULL);
        exe_time += (stop_time.tv_sec+(stop_time.tv_usec/1000000.0)) - (start_time.tv_sec+(start_time.tv_usec/1000000.0));
        }
        printf("\nMatrix Size : %d X %d : Executed 5 times : Average execution time for hipblasDgemv kernel is = %lf seconds.\n", N,N, exe_time/5);
        hipMemcpy(C_cublas,C_cublas_d,N*sizeof(double),hipMemcpyDeviceToHost);
//        print_vec(C_cublas);
        hipblasDestroy(handle);
        /////////////////////////////////////////////////////////////////////////
	/////////////////////////////////////////////////////////////////////////
        hipsparseHandle_t handle1;
        hipsparseSpMatDescr_t matA;
        hipsparseDnVecDescr_t Bd,Cspmvd;
        void* dBuffer = NULL;
        size_t bufferSize = 0;
        hipsparseCreate(&handle1);

        hipsparseCreateCsr(&matA, N, N, nnz, row_d, col_d, val_d, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);

        hipsparseCreateDnVec(&Bd, N, B_d, HIP_R_64F);

        hipsparseCreateDnVec(&Cspmvd, N, C_spmv_d, HIP_R_64F);

        hipsparseSpMV_bufferSize(handle1, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, Bd, &beta, Cspmvd, HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize);

        hipMalloc(&dBuffer, bufferSize);

	exe_time = 0.0;
	for(loop = 0; loop < 5; loop++)
	{
	gettimeofday(&start_time, NULL);
        hipsparseSpMV(handle1, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, Bd, &beta, Cspmvd, HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer);
	gettimeofday(&stop_time, NULL);
	exe_time += (stop_time.tv_sec+(stop_time.tv_usec/1000000.0)) - (start_time.tv_sec+(start_time.tv_usec/1000000.0));
	}
        printf("\nMatrix Size : %d X %d : Executed 5 times : Average execution time for hipsparseSpMV kernel is = %lf seconds.\n", N,N, exe_time/5);

        hipsparseDestroySpMat(matA);
        hipsparseDestroyDnVec(Bd);
        hipsparseDestroyDnVec(Cspmvd);
        hipsparseDestroy(handle1);

        hipMemcpy(C_spmv, C_spmv_d, N*sizeof(double), hipMemcpyDeviceToHost);
//        print_vec(C_spmv);
        ////////////////////////////////////////////////////////////////////////////
        ////////////////////////////////////////////////////////////////////////////
        
        error_routine(N, C_cuda, C_csr, C_cublas, C_spmv);
	
	hipFree(A_d);
        hipFree(B_d);
        hipFree(C_cuda_d);
        hipFree(C_csr_d);
        hipFree(C_cublas_d);
        hipFree(C_spmv_d);
        hipFree(val_d);
        hipFree(row_d);
        hipFree(col_d);
        hipFree(dBuffer);

	free(A);
        free(B);
        free(C_cuda);
        free(C_csr);
        free(C_cublas);
        free(C_spmv);
        free(val);
        free(row);
        free(col);
        
        return 0;
}
