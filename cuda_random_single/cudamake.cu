#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include<hipblas.h>
#include<sys/time.h>
#include"cudamake.h"

int main(int argc, char *argv[])
{
        double *A, *B, *C_csr, *C_spmv, *val;
        int *row, *col;
       	double *B_d, *C_csr_d, *C_spmv_d;
        int N = atoi(argv[1]);
        int nnz = N  * 5 / 100;
   	printf("%d\n", nnz);
	double exe_time;
	struct timeval stop_time, start_time;
	
        mat_vec_allocation(N, nnz, &A, &B, &C_csr, &C_spmv, &val, &row, &col); 
        
        sparse_mat_vec_initializer(N, nnz, A, B, val, col, row);
                
//        print_mat(N, A);
//        print_vec(N, B);

        double *val_d; 
        int *col_d, *row_d;
        
        hipMalloc(&B_d,N*sizeof(double));

        hipMalloc(&C_csr_d,N*sizeof(double));

        hipMalloc(&C_spmv_d,N*sizeof(double));

        hipMemcpy(B_d,B,N*sizeof(double),hipMemcpyHostToDevice);

	int Total_num_Threads = N;
        int num_threads_per_block = 256;
        int numblocks = Total_num_Threads/num_threads_per_block + 1;
        int  loop;
	//////////////////////////////////////////////////////////////////////

	///////////////////////////////////////////////////////////////////////
	///////////////////////////////////////////////////////////////////////
	

//	print_csr(N, val, row, col, nnz);
	
	hipMalloc(&val_d,N*nnz*sizeof(double));
        hipMalloc(&col_d,N*nnz*sizeof(int));
        hipMalloc(&row_d,(N+1)*sizeof(int));

        hipMemcpy(val_d,val,N*nnz*sizeof(double),hipMemcpyHostToDevice);
        hipMemcpy(col_d,col,N*nnz*sizeof(int),hipMemcpyHostToDevice);
        hipMemcpy(row_d,row,(N+1)*sizeof(int),hipMemcpyHostToDevice);
        
        exe_time = 0.0;
	for(loop = 0; loop < 5; loop++)
	{
        gettimeofday(&start_time, NULL);
	csr_multiplication<<<numblocks,num_threads_per_block>>>(N, B_d, C_csr_d, row_d, col_d, val_d);
	gettimeofday(&stop_time, NULL);
	exe_time += (stop_time.tv_sec+(stop_time.tv_usec/1000000.0)) - (start_time.tv_sec+(start_time.tv_usec/1000000.0));
	}
	printf("\nMatrix Size : %d X %d : Executed 5 times : Average execution time for csr mult kernel is = %lf seconds.\n", N,N, exe_time/5);
	hipMemcpy(C_csr, C_csr_d, N*sizeof(double), hipMemcpyDeviceToHost);
//	print_vec(N, C_csr);
	//////////////////////////////////////////////////////////////////////////
	/////////////////////////////////////////////////////////////////////////

        /////////////////////////////////////////////////////////////////////////
	/////////////////////////////////////////////////////////////////////////
	const double alpha = 1.0;
        const double beta = 0.0;
        
        hipsparseHandle_t handle1;
        hipsparseSpMatDescr_t matA;
        hipsparseDnVecDescr_t Bd,Cspmvd;
        void* dBuffer = NULL;
        size_t bufferSize = 0;
        hipsparseCreate(&handle1);

        hipsparseCreateCsr(&matA, N, N, N*nnz, row_d, col_d, val_d, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);

        hipsparseCreateDnVec(&Bd, N, B_d, HIP_R_64F);

        hipsparseCreateDnVec(&Cspmvd, N, C_spmv_d, HIP_R_64F);

        hipsparseSpMV_bufferSize(handle1, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, Bd, &beta, Cspmvd, HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize);

        hipMalloc(&dBuffer, bufferSize);

	exe_time = 0.0;
	for(loop = 0; loop < 5; loop++)
	{
	gettimeofday(&start_time, NULL);
        hipsparseSpMV(handle1, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, Bd, &beta, Cspmvd, HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer);
	gettimeofday(&stop_time, NULL);
	exe_time += (stop_time.tv_sec+(stop_time.tv_usec/1000000.0)) - (start_time.tv_sec+(start_time.tv_usec/1000000.0));
	}
        printf("\nMatrix Size : %d X %d : Executed 5 times : Average execution time for hipsparseSpMV kernel is = %lf seconds.\n", N,N, exe_time/5);

        hipsparseDestroySpMat(matA);
        hipsparseDestroyDnVec(Bd);
        hipsparseDestroyDnVec(Cspmvd);
        hipsparseDestroy(handle1);

        hipMemcpy(C_spmv, C_spmv_d, N*sizeof(double), hipMemcpyDeviceToHost);
//        print_vec(N, C_spmv);
        ////////////////////////////////////////////////////////////////////////////
        ////////////////////////////////////////////////////////////////////////////
        
        error_routine(N, C_csr, C_spmv);
	
        hipFree(B_d);

        hipFree(C_csr_d);

        hipFree(C_spmv_d);
        hipFree(val_d);
        hipFree(row_d);
        hipFree(col_d);
        hipFree(dBuffer);

	free(A);
        free(B);

        free(C_csr);

        free(C_spmv);
        free(val);
        free(row);
        free(col);
        
        return 0;
}
